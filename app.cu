
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel function
__global__ void add(int *d_array, int size)
{
    // Get the global index of the current thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we don't go out of bounds
    if (idx < size)
    {
        // Add 1 to the element at this index
        d_array[idx] += 1;
        // o_array[idx] = d_array[idx] + o_array[idx];
    }
}

int main()
{
    const int SIZE = 10;
    int h_array[SIZE]; // Host array
    int *d_array;      // Device array
    // int i_array[SIZE]; // Second Host array
    // int *e_array;      // Second Device array

    // Initialize the host array
    for (int i = 0; i < SIZE; i++)
    {
        h_array[i] = i;
        // i_array[i] = 256 - i;
        
    }

    // Allocate memory on the GPU
    hipMalloc((void **)&d_array, SIZE * sizeof(int));

    // Copy the host array to the GPU
    hipMemcpy(d_array, h_array, SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel (1 block, SIZE threads)
    add<<<1, SIZE>>>(d_array, SIZE);

    // Copy the result back to the host
    hipMemcpy(h_array, d_array, SIZE * sizeof(int), hipMemcpyDeviceToHost);

    // Print the results
    printf("Results:\n");
    for (int i = 0; i < SIZE; i++)
    {
        printf("%d ", h_array[i]);
    }
    printf("\n");

    // Free GPU memory
    hipFree(d_array);

    return 0;
}
